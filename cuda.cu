#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

__global__ void sprawdz_punkty(const double* xs, const double* ys, int* wynik, int n, double tolerancja) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        double x = xs[i];
        double y = ys[i];
        double y_calc = pow(x, 5) - pow(x, 2) + 2 * x + 1;
        wynik[i] = (fabs(y - y_calc) <= tolerancja) ? 1 : 0;
    }
}

int main() {
    std::ifstream file("punkty.txt");
    if (!file) {
        std::cerr << "Nie mogę otworzyć pliku punkty.txt\n";
        return 1;
    }

    std::vector<double> xs, ys;
    double x, y;
    while (file >> x >> y) {
        xs.push_back(x);
        ys.push_back(y);
    }

    int n = xs.size();
    double tolerancja = 0.001;

    double *d_xs, *d_ys;
    int *d_wynik, *wynik = new int[n];

    hipMalloc(&d_xs, n * sizeof(double));
    hipMalloc(&d_ys, n * sizeof(double));
    hipMalloc(&d_wynik, n * sizeof(int));

    hipMemcpy(d_xs, xs.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_ys, ys.data(), n * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Pomiar czasu
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    sprawdz_punkty<<<blocksPerGrid, threadsPerBlock>>>(d_xs, d_ys, d_wynik, n, tolerancja);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(wynik, d_wynik, n * sizeof(int), hipMemcpyDeviceToHost);

    int pasuje = 0, niepasuje = 0;
    for (int i = 0; i < n; ++i) {
        if (wynik[i]) pasuje++;
        else niepasuje++;
    }

    std::cout << "Pasuje: " << pasuje << "\nNie pasuje: " << niepasuje << "\n";
    std::cout << "Czas działania GPU: " << milliseconds << " ms\n";

    // Czyszczenie pamięci
    delete[] wynik;
    hipFree(d_xs);
    hipFree(d_ys);
    hipFree(d_wynik);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
